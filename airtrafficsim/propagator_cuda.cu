
#include <hip/hip_runtime.h>
#include <math.h>

#define DEG2RAD 0.0174532925
#define RAD2DEG 57.2957795
#define FT_PER_MIN_TO_M_PER_SEC 0.00508

// nvcc -Xcompiler -fPIC -shared propagator_cuda.cu -o propagator_cuda.so


__device__
inline size_t idx(size_t i, size_t y, size_t vec_len=3){

    /* The numpy vectors are flattened for some reason. This fcn gives the correct index. */

    return i*vec_len + y;
}


__global__
void propagate(int n, float *position, float *velocity, float dt,
               float *turn_rate, float *climb_rate, float *heading){
    
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
        
    for (int i = index; i < n; i += stride){
        float horizontal_speed = sqrt(velocity[idx(i,0)]*velocity[idx(i,0)] + 
                                    velocity[idx(i,1)]*velocity[idx(i,1)]);
        heading[i] = atan2(velocity[idx(i,1)], velocity[idx(i,0)])
                                    + turn_rate[i] * dt * DEG2RAD;

        velocity[idx(i, 0)] = cos(heading[i]) * horizontal_speed;
        velocity[idx(i, 1)] = sin(heading[i]) * horizontal_speed;
        velocity[idx(i, 2)] = climb_rate[i] * FT_PER_MIN_TO_M_PER_SEC;

        position[idx(i, 0)] += velocity[idx(i, 0)] * dt;
        position[idx(i, 1)] += velocity[idx(i, 1)] * dt;
        position[idx(i, 2)] += velocity[idx(i, 2)] * dt;
    }
}


extern "C" void propagate_cuda(float *position,
                               float *velocity,
                               float dt,
                               float *turn_rate,
                               float *climb_rate,
                               float *heading,
                               int fleet_size){
    
    float *d_position, *d_velocity, *d_turn_rate, *d_climb_rate, *d_heading;
    
    hipMalloc(&d_position, fleet_size*3*sizeof(float));
    hipMalloc(&d_velocity, fleet_size*3*sizeof(float));
    hipMalloc(&d_turn_rate, fleet_size*sizeof(float));
    hipMalloc(&d_climb_rate, fleet_size*sizeof(float));
    hipMalloc(&d_heading, fleet_size*sizeof(float));
    
    hipMemcpy(d_position, position, fleet_size*3*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_velocity, velocity, fleet_size*3*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_turn_rate, turn_rate, fleet_size*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_climb_rate, climb_rate, fleet_size*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_heading, heading, fleet_size*sizeof(float), hipMemcpyHostToDevice);


    propagate<<<1, 1>>>(fleet_size, d_position, d_velocity, dt, d_turn_rate,
                        d_climb_rate, d_heading); 

    hipDeviceSynchronize();
        
    hipMemcpy(position, d_position, fleet_size*3*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(velocity, d_velocity, fleet_size*3*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(turn_rate, d_turn_rate, fleet_size*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(climb_rate, d_climb_rate, fleet_size*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(heading, d_heading, fleet_size*sizeof(float), hipMemcpyDeviceToHost);
    
    hipFree(position);
    hipFree(velocity);
    hipFree(turn_rate);
    hipFree(climb_rate);
    hipFree(heading); 
    return;
}
